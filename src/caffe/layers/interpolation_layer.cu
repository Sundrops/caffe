#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/interpolation_layer.hpp"
#include "caffe/util/math_functions.hpp"

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))



namespace caffe {

template <typename Dtype>
__global__ void InflateForwardGPU(const int nthreads,
          const Dtype* bottom_data, const int bottom_height, const int bottom_width, 
          Dtype *top_data, const int top_height, const int top_width, 
          const float factor_h, const float factor_w) {
          
    
          
    CUDA_KERNEL_LOOP(index, nthreads) {
    
        const int margin = 1;
        
        // index refers to to top_data
        const int y_t = index / top_width;
        const int x_t = index % top_width;
        
        
        // coordinate on target map
        const int idx_t = y_t * top_width + x_t;
            
        top_data[idx_t] = 0;
            
        float y_s = y_t / factor_h;
        float x_s = x_t / factor_w;
            
        for (int n = MAX(floor(y_s - margin) + 1, 0); n < MIN(y_s + margin, bottom_height); n++) {
            for (int m = MAX(floor(x_s - margin) + 1, 0); m < MIN(x_s + margin, bottom_width); m++) {
             
                top_data[idx_t] += bottom_data[n * bottom_width + m] * (margin - abs(x_s - m)) * (margin - abs(y_s - n));

            }
        }
        
    }
}





template <typename Dtype>
void InterpolationLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    // get parameters
    const int num = bottom[0]->num();
    const int channels = bottom[0]->channels();
    const int height = bottom[0]->height();
    const int width = bottom[0]->width();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    
    // new shape
    const int top_height = top[0]->height();
    const int top_width = top[0]->width();

    // resize
    const int nthreads = top_height * top_width;
    for (int n = 0; n < num; n++) {
        for (int c = 0; c < channels; c++) {    
      
            const int index_in = (n * channels + c) * height * width;
            const int index_out = (n * channels + c) * top_height * top_width;

            InflateForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(nthreads, bottom_data + index_in, height, width, top_data + index_out, top_height, top_width, factor_value_h_, factor_value_w_);

        }

    }
}




template <typename Dtype>
__global__ void InflateBackwardGPU(const int nthreads, 
            Dtype *bottom_diff, const int bottom_height, const int bottom_width, 
            const Dtype *top_diff, const int top_height, const int top_width, 
            const float factor_h, const float factor_w) {
            

    const float normalizer = factor_h * factor_w;

    CUDA_KERNEL_LOOP(index, nthreads) {
    
        const int margin = 1;
        
        // index refers to to top_data
        const int n = index / bottom_width;
        const int m = index % bottom_width;
        
        const int idx_s = n * bottom_width + m;
        bottom_diff[idx_s] = 0;
        
        for (int y_t = MAX(floor((n - margin) * factor_h) + 1, 0); y_t < MIN((n + margin) * factor_h, top_height); y_t++) {
            for (int x_t = MAX(floor((m - margin) * factor_w) + 1, 0); x_t < MIN((m + margin) * factor_w, top_width); x_t++) {
                
                // diff
                bottom_diff[idx_s] += top_diff[y_t * top_width + x_t] 
                                      * (margin - abs((x_t / factor_w) - m)) * (margin - abs((y_t / factor_h) - n));
            }
        }
        
        bottom_diff[idx_s] /= normalizer;
    }
}





template <typename Dtype>
void InterpolationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    
    // get parameters
    const int num = bottom[0]->num();
    const int channels = bottom[0]->channels();
    const int height = bottom[0]->height();
    const int width = bottom[0]->width();
    const int top_height = top[0]->height();
    const int top_width = top[0]->width();
    
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* top_diff = top[0]->gpu_diff();



    if (propagate_down[0]) {

        // compute diff for bottom
        const int nthreads = height * width;
        
        for (int n = 0; n < num; n++) {
            for (int c = 0; c < channels; c++) {
                const int index_in = (n * channels + c) * height * width;
                const int index_out = (n * channels + c) * top_height * top_width;
                InflateBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(nthreads, bottom_diff + index_in, height, width, top_diff + index_out, top_height, top_width, factor_value_h_, factor_value_w_);
            }
        }
    }
        
    
    if (propagate_down[1]) {
        LOG(FATAL) << this->type()
                   << " Layer cannot backpropagate to the second input.";
    }

    
}

INSTANTIATE_LAYER_GPU_FUNCS(InterpolationLayer);

}  // namespace caffe
