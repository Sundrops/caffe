#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include "caffe/layers/remap_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {







template <typename Dtype>
__global__ void RemapForwardGPU(const int nthreads,
          const Dtype* Vb,
          const Dtype* coords,
          Dtype *Vt, 
          const int H1, const int W1) {
          
    CUDA_KERNEL_LOOP(index, nthreads) {
    
        const int h = index / W1;
        const int w = index % W1;
        
        float x, y, wx, wy, w00, w01, w10, w11, v00, v01, v10, v11;
        int x0, y0, x1, y1;
        
        x = w + coords[h*W1+w];
		y = h + coords[(H1+h)*W1+w];  // from 1 to 2
										
		x0 = floor(x);
		y0 = floor(y);
		x1 = x0 + 1;
		y1 = y0 + 1;
		wx = x - x0;
		wy = y - y0;
		w00 = (1 - wx) * (1 - wy);
		w01 = (1 - wx) * wy;
		w10 = wx * (1 - wy);
		w11 = wx * wy;
		
		v00 = (x0 < 0 || x0 > W1 - 1 || y0 < 0 || y0 > H1 - 1) ? 0 : Vb[y0*W1+x0];	
		v01 = (x0 < 0 || x0 > W1 - 1 || y1 < 0 || y1 > H1 - 1) ? 0 : Vb[y1*W1+x0];		
		v10 = (x1 < 0 || x1 > W1 - 1 || y0 < 0 || y0 > H1 - 1) ? 0 : Vb[y0*W1+x1];		
		v11 = (x1 < 0 || x1 > W1 - 1 || y1 < 0 || y1 > H1 - 1) ? 0 : Vb[y1*W1+x1];
		
		Vt[h*W1+w] = w00 * v00 + w01 * v01 + w10 * v10 + w11 * v11;

    }
}



template <typename Dtype>
void RemapLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    
	const Dtype* Vb = bottom[0]->gpu_data(); // image2
	const Dtype* coords = bottom[1]->gpu_data();
	Dtype* Vt = top[0]->mutable_gpu_data();
	int N  = bottom[0]->num();
	int C  = bottom[0]->channels();
	int W0 = bottom[0]->width();
	int H0 = bottom[0]->height();
	int W1 = bottom[1]->width();
	int H1 = bottom[1]->height();
	
	CHECK_EQ(W0, W1);
	CHECK_EQ(H0, H1);
	
	const int nthreads = H1 * W1;
	for ( int n = 0; n < N; n++ ) {
		for ( int c = 0; c < C; c++ ) {
		
		    const int idx_map = (n * C + c) * H1 * W1;
		    const int idx_coord = (n * 2) * H1 * W1;
		    
		    RemapForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(nthreads, Vb + idx_map, coords + idx_coord,  Vt + idx_map, H1, W1);

		}
	} 


}







template <typename Dtype>
void RemapLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {


    Backward_cpu(top, propagate_down, bottom);
   
   
}



INSTANTIATE_LAYER_GPU_FUNCS(RemapLayer);


}  // namespace caffe
